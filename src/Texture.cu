#include "GLCUDA.h"
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

void fillIntBuffer(hipGraphicsResource *resource, int val)
{
	cuChk(hipGraphicsMapResources(1, &resource, NULL));
	int* devPtr;
	size_t  size;
	cuChk(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource));
	thrust::device_ptr<int> t_devPtr = thrust::device_pointer_cast(devPtr);

	int pointSize = sizeof(int);
	int numPoints = size / pointSize;

	thrust::fill(t_devPtr, t_devPtr + numPoints, val);

	cuChk(hipGraphicsUnmapResources(1, &resource, NULL));
}

void fillIntTexture(hipGraphicsResource *resource, int width, int height, int val)
{
	cuChk(hipGraphicsMapResources(1, &resource, NULL));
	hipArray_t arr;
	cuChk(hipGraphicsSubResourceGetMappedArray(&arr, resource, 0, 0));

	thrust::device_vector<int> buffer(width*height);
	thrust::fill(buffer.begin(), buffer.end(), val);
	
	int *src = buffer.data().get();
	int spitch = sizeof(int) * width; // number of bytes of a row
	cuChk(hipMemcpy2DToArray(arr, 0, 0, src, spitch, width, height, hipMemcpyDeviceToDevice));

	cuChk(hipGraphicsUnmapResources(1, &resource, NULL));
}
